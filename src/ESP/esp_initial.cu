#include "hip/hip_runtime.h"
// ==============================================================================
// This file is part of THOR.
//
//     THOR is free software : you can redistribute it and / or modify
//     it under the terms of the GNU General Public License as published by
//     the Free Software Foundation, either version 3 of the License, or
//     (at your option) any later version.
//
//     THOR is distributed in the hope that it will be useful,
//     but WITHOUT ANY WARRANTY; without even the implied warranty of
//     MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.See the
//     GNU General Public License for more details.
//
//     You find a copy of the GNU General Public License in the main
//     THOR directory under <license.txt>.If not, see
//     <http://www.gnu.org/licenses/>.
// ==============================================================================
//
// Build the class ESP (Exoclimes Simulation Platform)
//
//
// Description:
//   Declare and initialize variables in the model
//
// Method: -
//
//
// Known limitations: None.
//
//
// Known issues: None.
//
//
// If you use this code please cite the following reference:
//
//       [1] Mendonca, J.M., Grimm, S.L., Grosheintz, L., & Heng, K., ApJ, 829, 115, 2016
//
// Current Code Owners: Joao Mendonca (joao.mendonca@space.dtu.dk)
//                      Russell Deitrick (russell.deitrick@csh.unibe.ch)
//                      Urs Schroffenegger (urs.schroffenegger@csh.unibe.ch)
//
// History:
// Version Date       Comment
// ======= ====       =======
// 2.0     30/11/2018 Released version (RD & US)
// 1.0     16/08/2017 Released version  (JM)
//
////////////////////////////////////////////////////////////////////////

#include "directories.h"
#include "esp.h"
#include "log_writer.h"
#include "phy/profx_globdiag.h"
#include "phy/ultrahot_thermo.h"
#include "phy/valkyrie_jet_steadystate.h"
#include "storage.h"

#include <map>
#include <stdio.h>

// physical modules
#include "phy_modules.h"

__host__ ESP::ESP(int *                 point_local_,
                  int *                 maps_,
                  double *              lonlat_,
                  double *              Altitude_,
                  double *              Altitudeh_,
                  double *              nvecoa_,
                  double *              nvecti_,
                  double *              nvecte_,
                  double *              areasT_,
                  double *              areasTr_,
                  double *              areas_,
                  double *              div_,
                  double *              grad_,
                  double *              curlz_,
                  double *              func_r_,
                  int                   nl_region_,
                  int                   nr_,
                  int                   nv_,
                  int                   nvi_,
                  int                   glevel_,
                  bool                  spring_dynamics_,
                  double                spring_beta_,
                  int                   nlat_bins_,
                  int *                 zonal_mean_tab,
                  double                Ruv_sponge_,
                  double                Rw_sponge_,
                  double                RT_sponge_,
                  double                ns_ray_sponge_,
                  bool                  damp_uv_to_mean_,
                  bool                  damp_w_to_mean_,
                  raysp_calc_mode_types raysp_calc_mode_,
                  double                Dv_sponge_,
                  double                ns_diff_sponge_,
                  int                   order_diff_sponge_,
                  double                t_shrink_,
                  bool                  shrink_sponge_,
                  int                   point_num_,
                  bool                  globdiag,
                  benchmark_types       core_benchmark_,
                  log_writer &          logwriter_,
                  int                   max_count_,
                  bool                  output_mean,
                  init_PT_profile_types init_PT_profile_,
                  double                Tint_,
                  double                kappa_lw_,
                  double                kappa_sw_,
                  double                f_lw_,
                  double                bv_freq_,
                  uh_thermo_types       ultrahot_thermo_,
                  uh_heating_types      ultrahot_heating_,
                  thermo_equation_types thermo_equation_) :
    nl_region(nl_region_),
    nr(nr_),
    point_num(point_num_),
    nv(nv_),
    nvi(nvi_),
    nlat_bins(nlat_bins_),
    order_diff_sponge(order_diff_sponge_),
    damp_uv_to_mean(damp_uv_to_mean_),
    damp_w_to_mean(damp_w_to_mean_),
    glevel(glevel_),
    spring_dynamics(spring_dynamics_),
    spring_beta(spring_beta_),
    logwriter(logwriter_),
    core_benchmark(core_benchmark_),
    init_PT_profile(init_PT_profile_),
    raysp_calc_mode(raysp_calc_mode_),
    ultrahot_thermo(ultrahot_thermo_),
    ultrahot_heating(ultrahot_heating_),
    thermo_equation(thermo_equation_),
    shrink_sponge(shrink_sponge_) {

    point_local_h = point_local_;
    maps_h        = maps_;

    lonlat_h = lonlat_;

    Altitude_h  = Altitude_;
    Altitudeh_h = Altitudeh_;

    nvecoa_h  = nvecoa_;
    nvecti_h  = nvecti_;
    nvecte_h  = nvecte_;
    areasTr_h = areasTr_;
    areasT_h  = areasT_;
    areas_h   = areas_;

    div_h   = div_;
    grad_h  = grad_;
    curlz_h = curlz_;

    func_r_h = func_r_;

    zonal_mean_tab_h = zonal_mean_tab;

    Ruv_sponge     = Ruv_sponge_;
    Rw_sponge      = Rw_sponge_;
    RT_sponge      = RT_sponge_;
    ns_ray_sponge  = ns_ray_sponge_;
    Dv_sponge      = Dv_sponge_;
    ns_diff_sponge = ns_diff_sponge_;

    t_shrink  = t_shrink_;
    max_count = max_count_;

    Tint     = Tint_;
    kappa_lw = kappa_lw_;
    kappa_sw = kappa_sw_;
    f_lw     = f_lw_;
    bv_freq  = bv_freq_;

    // Set the physics module execute state for the rest of the lifetime of ESP object
    // only execute physics modules when no benchmarks are enabled
    if (core_benchmark == NO_BENCHMARK) {
        phy_modules_execute = true;
    }
    else
        phy_modules_execute = false;

    //
    //  Allocate Data
    alloc_data(globdiag, output_mean);
}

__host__ void ESP::alloc_data(bool globdiag, bool output_mean) {

    //
    //  Description:
    //
    //  Allocate data on host and device.
    //
    //  Allocate data in host
    //  Diagnostics an doutput
    Rho_h         = (double *)malloc(nv * point_num * sizeof(double));
    pressure_h    = (double *)malloc(nv * point_num * sizeof(double));
    temperature_h = (double *)malloc(nv * point_num * sizeof(double));
    Mh_h          = (double *)malloc(nv * point_num * 3 * sizeof(double));
    W_h           = (double *)malloc(nv * point_num * sizeof(double));
    Wh_h          = (double *)malloc(nvi * point_num * sizeof(double));

    if (output_mean == true) {
        Rho_mean_h      = (double *)malloc(nv * point_num * sizeof(double));
        pressure_mean_h = (double *)malloc(nv * point_num * sizeof(double));
        Mh_mean_h       = (double *)malloc(nv * point_num * 3 * sizeof(double));
        Wh_mean_h       = (double *)malloc(nvi * point_num * sizeof(double));
    }

    Etotal_h  = (double *)malloc(nv * point_num * sizeof(double));
    Mass_h    = (double *)malloc(nv * point_num * sizeof(double));
    AngMomx_h = (double *)malloc(nv * point_num * sizeof(double));
    AngMomy_h = (double *)malloc(nv * point_num * sizeof(double));
    AngMomz_h = (double *)malloc(nv * point_num * sizeof(double));
    Entropy_h = (double *)malloc(nv * point_num * sizeof(double));

    // ultra-hot jupiter stuff
    Rd_h = (double *)malloc(nv * point_num * sizeof(double));
    Cp_h = (double *)malloc(nv * point_num * sizeof(double));

    flux_vec        = (double *)malloc(nv * point_num * sizeof(double));
    boundary_flux_h = (double *)malloc(6 * nv * point_num * sizeof(double));
    hipMalloc((void **)&boundary_flux_d, 6 * point_num * nv * sizeof(double));

    //  Allocate data in device
    //  Grid
    hipMalloc((void **)&point_local_d, 6 * point_num * sizeof(int));
    hipMalloc((void **)&maps_d, (nl_region + 2) * (nl_region + 2) * nr * sizeof(int));

    //  Operators
    hipMalloc((void **)&nvecoa_d, 6 * 3 * point_num * sizeof(double));
    hipMalloc((void **)&nvecti_d, 6 * 3 * point_num * sizeof(double));
    hipMalloc((void **)&nvecte_d, 6 * 3 * point_num * sizeof(double));
    hipMalloc((void **)&areasT_d, point_num * sizeof(double));
    hipMalloc((void **)&areasTr_d, 6 * point_num * sizeof(double));
    hipMalloc((void **)&areas_d, 3 * 6 * point_num * sizeof(double));
    hipMalloc((void **)&func_r_d, 3 * point_num * sizeof(double));
    hipMalloc((void **)&div_d, 7 * 3 * point_num * sizeof(double));
    hipMalloc((void **)&grad_d, 7 * 3 * point_num * sizeof(double));

    //  Altitude (grid)
    hipMalloc((void **)&Altitude_d, nv * sizeof(double));
    hipMalloc((void **)&Altitudeh_d, nvi * sizeof(double));

    //  Longitude-latitude
    hipMalloc((void **)&lonlat_d, 2 * point_num * sizeof(double));

    //  Diagnostics
    hipMalloc((void **)&Mh_d, nv * point_num * 3 * sizeof(double));
    hipMalloc((void **)&W_d, nv * point_num * sizeof(double));
    hipMalloc((void **)&Wh_d, nvi * point_num * sizeof(double));
    hipMalloc((void **)&Rho_d, nv * point_num * sizeof(double));
    hipMalloc((void **)&pressure_d, nv * point_num * sizeof(double));
    hipMalloc((void **)&pressureh_d, (nv + 1) * point_num * sizeof(double));

    if (output_mean == true) {
        // Average quantities over output interval
        hipMalloc((void **)&Mh_mean_d, nv * point_num * 3 * sizeof(double));
        hipMalloc((void **)&Wh_mean_d, nvi * point_num * sizeof(double));
        hipMalloc((void **)&Rho_mean_d, nv * point_num * sizeof(double));
        hipMalloc((void **)&pressure_mean_d, nv * point_num * sizeof(double));
    }

    // ultra hot
    hipMalloc((void **)&Rd_d, nv * point_num * sizeof(double));
    hipMalloc((void **)&Cp_d, nv * point_num * sizeof(double));

    //  Temperature
    hipMalloc((void **)&temperature_d, nv * point_num * sizeof(double));

    //  Potential temperature
    hipMalloc((void **)&pt_d, nv * point_num * sizeof(double));
    hipMalloc((void **)&pth_d, nvi * point_num * sizeof(double));

    //  Energy (for thermo_equation = energy)
    hipMalloc((void **)&epotential_d, nv * point_num * sizeof(double));
    hipMalloc((void **)&epotentialh_d, nvi * point_num * sizeof(double));
    hipMalloc((void **)&ekinetic_d, nv * point_num * sizeof(double));
    hipMalloc((void **)&ekinetich_d, nvi * point_num * sizeof(double));
    hipMalloc((void **)&Etotal_tau_d, nv * point_num * sizeof(double));

    //  Entalphy
    hipMalloc((void **)&h_d, nv * point_num * sizeof(double));
    hipMalloc((void **)&hh_d, nvi * point_num * sizeof(double));

    //  Advection
    hipMalloc((void **)&Adv_d, nv * point_num * 3 * sizeof(double));

    //  3D vector
    hipMalloc((void **)&v_d, nv * point_num * 3 * sizeof(double));

    //  Effective gravity
    hipMalloc((void **)&gtil_d, nv * point_num * sizeof(double));
    hipMalloc((void **)&gtilh_d, nvi * point_num * sizeof(double));

    //  Slow modes
    hipMalloc((void **)&SlowMh_d, nv * point_num * 3 * sizeof(double));
    hipMalloc((void **)&SlowWh_d, nvi * point_num * sizeof(double));
    hipMalloc((void **)&SlowRho_d, nv * point_num * sizeof(double));
    hipMalloc((void **)&Slowpressure_d, nv * point_num * sizeof(double));


    //  Deviations
    hipMalloc((void **)&pressures_d, nv * point_num * sizeof(double));
    hipMalloc((void **)&Rhos_d, nv * point_num * sizeof(double));
    hipMalloc((void **)&Mhs_d, nv * point_num * 3 * sizeof(double));
    hipMalloc((void **)&Ws_d, nv * point_num * sizeof(double));
    hipMalloc((void **)&Whs_d, nvi * point_num * sizeof(double));


    //  RK-Method
    hipMalloc((void **)&pressurek_d, nv * point_num * sizeof(double));
    hipMalloc((void **)&Rhok_d, nv * point_num * sizeof(double));
    hipMalloc((void **)&Mhk_d, nv * point_num * 3 * sizeof(double));
    hipMalloc((void **)&Wk_d, nv * point_num * sizeof(double));
    hipMalloc((void **)&Whk_d, nvi * point_num * sizeof(double));

    //  Vertical integration
    hipMalloc((void **)&Sp_d, nv * point_num * sizeof(double));
    hipMalloc((void **)&Sd_d, nv * point_num * sizeof(double));

    //  Diffusion
    hipMalloc((void **)&Kdhz_d, nv * sizeof(double));
    hipMalloc((void **)&Kdh4_d, nv * sizeof(double));
    hipMalloc((void **)&Kdvz_d, nv * sizeof(double));
    hipMalloc((void **)&Kdv6_d, nv * sizeof(double));

    hipMalloc((void **)&DivM_d, nv * point_num * 3 * sizeof(double));
    hipMalloc((void **)&diffpr_d, nv * point_num * sizeof(double));
    hipMalloc((void **)&diffmh_d, 3 * nv * point_num * sizeof(double));
    hipMalloc((void **)&diffw_d, nv * point_num * sizeof(double));
    hipMalloc((void **)&diffrh_d, nv * point_num * sizeof(double));
    hipMalloc((void **)&diff_d, 6 * nv * point_num * sizeof(double));
    hipMalloc((void **)&divg_Mh_d, 3 * nv * point_num * sizeof(double));

    hipMalloc((void **)&Kdh2_d, nv * sizeof(double));

    hipMalloc((void **)&diffprv_d, nv * point_num * sizeof(double));
    hipMalloc((void **)&diffmv_d, 3 * nv * point_num * sizeof(double));
    hipMalloc((void **)&diffwv_d, nv * point_num * sizeof(double));
    hipMalloc((void **)&diffrv_d, nv * point_num * sizeof(double));
    hipMalloc((void **)&diffv_d1, 6 * nv * point_num * sizeof(double));
    hipMalloc((void **)&diffv_d2, 6 * nv * point_num * sizeof(double));

    
    profx_Qheat_h = (double *)malloc(nv * point_num * sizeof(double));
    
    hipMalloc((void **)&profx_Qheat_d, nv * point_num * sizeof(double));
    hipMalloc((void **)&profx_dMh_d, 3 * nv * point_num * sizeof(double));
    hipMalloc((void **)&profx_dWh_d, nvi * point_num * sizeof(double));
    hipMalloc((void **)&profx_dW_d, nv * point_num * sizeof(double));

    //  Extras-nan
    hipMalloc((void **)&check_d, sizeof(bool));

    hipMalloc((void **)&vbar_d, 3 * nv * nlat_bins * sizeof(double));
    hipMalloc((void **)&zonal_mean_tab_d, 3 * point_num * sizeof(int));
    vbar_h = (double *)malloc(3 * nv * nlat_bins * sizeof(double));
    hipMalloc((void **)&utmp, nv * nlat_bins * max_count * sizeof(double));
    hipMalloc((void **)&vtmp, nv * nlat_bins * max_count * sizeof(double));
    hipMalloc((void **)&wtmp, nv * nlat_bins * max_count * sizeof(double));
    utmp_h = (double *)malloc(nv * nlat_bins * max_count * sizeof(double));
    vtmp_h = (double *)malloc(nv * nlat_bins * max_count * sizeof(double));
    wtmp_h = (double *)malloc(nv * nlat_bins * max_count * sizeof(double));
    hipMalloc((void **)&Tbar_d, nv * nlat_bins * sizeof(double));
    Tbar_h = (double *)malloc(nv * nlat_bins * sizeof(double));
    hipMalloc((void **)&Ttmp, nv * nlat_bins * max_count * sizeof(double));
    Ttmp_h = (double *)malloc(nv * nlat_bins * max_count * sizeof(double));

    if (globdiag == true) {
        //  globdiag quantities
        hipMalloc((void **)&Etotal_d, nv * point_num * sizeof(double));
        hipMalloc((void **)&Entropy_d, nv * point_num * sizeof(double));
        hipMalloc((void **)&Mass_d, nv * point_num * sizeof(double));
        hipMalloc((void **)&AngMomx_d, nv * point_num * sizeof(double));
        hipMalloc((void **)&AngMomy_d, nv * point_num * sizeof(double));
        hipMalloc((void **)&AngMomz_d, nv * point_num * sizeof(double));
        hipMalloc((void **)&GlobalE_d, 1 * sizeof(double));
        hipMalloc((void **)&GlobalEnt_d, 1 * sizeof(double));
        hipMalloc((void **)&GlobalMass_d, 1 * sizeof(double));
        hipMalloc((void **)&GlobalAMx_d, 1 * sizeof(double));
        hipMalloc((void **)&GlobalAMy_d, 1 * sizeof(double));
        hipMalloc((void **)&GlobalAMz_d, 1 * sizeof(double));
    }

    // PHY modules
    log::printf("  Dynamical core memory initialised.\n");

    if (phy_modules_execute) {

        // physics module need to initialise their own memory
        bool init_modules = phy_modules_init_mem(*this, phy_modules_core_arrays);
        // Physics module register arrays that need to be updated in dynamical core Runge-Kutta step
        phy_modules_core_arrays.allocate_device_array();
        if (init_modules)
            log::printf("  Module memory initialised.\n");
        else {
            log::printf("  Error initialising module memory.\n");
            exit(-1);
        }
    }
}

__host__ bool ESP::initial_values(const std::string &initial_conditions_filename,
                                  const bool &       continue_sim,
                                  double             timestep_dyn,
                                  SimulationSetup &  sim,
                                  int &              nstep,
                                  double &           simulation_start_time,
                                  int &              output_file_idx) {

    output_file_idx = 0;
    nstep           = 0;
    //  Set initial conditions.
    //
    //
    //  Initial atmospheric conditions
    bool   read_gibbs = read_in_gibbs_H(GibbsN); //ultrahot jup
    double chi_H = 0, ptmp, eps = 1e-8, f, df, dz, mu;
    int    it, it_max = 100;

    double Rd_L, P_L, T_L, rho_L, alpha;
    if (sim.rest) {
        for (int i = 0; i < point_num; i++) {
            //
            //          Initial conditions for an isothermal Atmosphere
            //
            if ((init_PT_profile == ISOTHERMAL || init_PT_profile == CONSTBV)
                && ultrahot_thermo == NO_UH_THERMO) {
                //isothermal initial profile, no variation in Rd or Cp due to H-H2 reaction
                //exact solution to hydrostatic equation
                // double Ha = sim.Rd * sim.Tmean / sim.Gravit;
                // for (int lev = 0; lev < nv; lev++) {
                //     pressure_h[i * nv + lev]    = sim.P_Ref * exp(-Altitude_h[lev] / Ha);
                //     temperature_h[i * nv + lev] = sim.Tmean;
                //     Rd_h[i * nv + lev]          = sim.Rd;
                //     Cp_h[i * nv + lev]          = sim.Cp;
                // }

                //iterative solution to hydrostatic equation
                for (int lev = 0; lev < nv; lev++) {
                    //first, we define thermo quantities of layer below and make
                    //our initial guess for the Newton-Raphson solver
                    if (lev == 0) {
                        P_L   = sim.P_Ref;
                        rho_L = sim.P_Ref / (sim.Rd * sim.Tmean);
                        T_L   = sim.Tmean;
                        dz    = Altitude_h[0];
                    }
                    else {
                        P_L   = pressure_h[i * nv + lev - 1];
                        rho_L = Rho_h[i * nv + lev - 1];
                        T_L   = temperature_h[i * nv + lev - 1];
                        dz    = Altitude_h[lev] - Altitude_h[lev - 1];
                    }
                    pressure_h[i * nv + lev]    = P_L;
                    Rd_h[i * nv + lev]          = sim.Rd;
                    Cp_h[i * nv + lev]          = sim.Cp;
                    Rho_h[i * nv + lev]         = rho_L;
                    temperature_h[i * nv + lev] = T_L;
                    ptmp                        = pressure_h[i * nv + lev] + 2 * eps;

                    it = 0;
                    while (it < it_max && ptmp - pressure_h[i * nv + lev] > eps) {
                        //Newton-Raphson solver of hydrostatic eqn for thermo properties
                        ptmp = pressure_h[i * nv + lev];
                        f    = (pressure_h[i * nv + lev] - P_L) / dz
                            + sim.Gravit * 0.5 * (Rho_h[i * nv + lev] + rho_L);
                        df = 1.0 / dz + 0.5 * sim.Gravit / (sim.Rd * temperature_h[i * nv + lev]);
                        pressure_h[i * nv + lev] = pressure_h[i * nv + lev] - f / df;
                        if (init_PT_profile == CONSTBV) {
                            //use a constant brunt-vaisala freq
                            //alpha is a function equal to 1/2*(1/T)*dT
                            alpha = 0.5 * pow(bv_freq, 2) / sim.Gravit * dz
                                    + sim.Rd / sim.Cp * (pressure_h[i * nv + lev] - P_L)
                                          / (pressure_h[i * nv + lev] + P_L);
                            temperature_h[i * nv + lev] = (1 + alpha) * T_L / (1 - alpha);
                        }
                        Rho_h[i * nv + lev] =
                            pressure_h[i * nv + lev] / (sim.Rd * temperature_h[i * nv + lev]);

                        it++;
                    }
                }
            }
            else {
                //
                //          Initial conditions for a non-isothermal Atmosphere
                //
                mu = 0.5;

                for (int lev = 0; lev < nv; lev++) {
                    //first, we define thermo quantities of layer below and make
                    //our initial guess for the Newton-Raphson solver
                    if (lev == 0) {
                        if (init_PT_profile == ISOTHERMAL) {
                            temperature_h[i * nv + lev] = sim.Tmean;
                        }
                        else {
                            temperature_h[i * nv + lev] = guillot_T(sim.P_Ref,
                                                                    mu,
                                                                    sim.Tmean,
                                                                    sim.P_Ref,
                                                                    sim.Gravit,
                                                                    Tint,
                                                                    f_lw,
                                                                    kappa_sw,
                                                                    kappa_lw);
                        }
                        if (ultrahot_thermo != NO_UH_THERMO) {
                            chi_H = chi_H_equilibrium(
                                GibbsT, GibbsdG, GibbsN, temperature_h[i * nv + lev], sim.P_Ref);
                            Rd_L = Rd_from_chi_H(chi_H);
                        }
                        else {
                            Rd_L = sim.Rd;
                        }
                        P_L = sim.P_Ref;
                        T_L = temperature_h[i * nv + lev];
                        dz  = Altitude_h[0];
                    }
                    else {
                        temperature_h[i * nv + lev] = temperature_h[i * nv + lev - 1];
                        if (ultrahot_thermo != NO_UH_THERMO) {
                            chi_H = chi_H_equilibrium(
                                GibbsT, GibbsdG, GibbsN, sim.Tmean, pressure_h[i * nv + lev - 1]);
                            Rd_L = Rd_h[i * nv + lev - 1];
                        }
                        else {
                            Rd_L = Rd_h[i * nv + lev - 1];
                        }
                        P_L = pressure_h[i * nv + lev - 1];
                        T_L = temperature_h[i * nv + lev - 1];
                        dz  = Altitude_h[lev] - Altitude_h[lev - 1];
                    }
                    pressure_h[i * nv + lev] = P_L;
                    Rd_h[i * nv + lev]       = Rd_L;
                    ptmp                     = pressure_h[i * nv + lev] + 2 * eps;

                    it = 0;
                    while (it < it_max && ptmp - pressure_h[i * nv + lev] > eps) {
                        //Newton-Raphson solver of hydrostatic eqn for thermo properties
                        ptmp = pressure_h[i * nv + lev];
                        f    = log(pressure_h[i * nv + lev] / P_L) / dz
                            + sim.Gravit
                                  / (0.5
                                     * (Rd_h[i * nv + lev] * temperature_h[i * nv + lev]
                                        + Rd_L * T_L));
                        df                       = 1.0 / (pressure_h[i * nv + lev] * dz);
                        pressure_h[i * nv + lev] = pressure_h[i * nv + lev] - f / df;
                        if (init_PT_profile == ISOTHERMAL) {
                            temperature_h[i * nv + lev] = sim.Tmean;
                        }
                        else {
                            temperature_h[i * nv + lev] = guillot_T(pressure_h[i * nv + lev],
                                                                    mu,
                                                                    sim.Tmean,
                                                                    sim.P_Ref,
                                                                    sim.Gravit,
                                                                    Tint,
                                                                    f_lw,
                                                                    kappa_sw,
                                                                    kappa_lw);
                        }
                        if (ultrahot_thermo != NO_UH_THERMO) {
                            chi_H              = chi_H_equilibrium(GibbsT,
                                                      GibbsdG,
                                                      GibbsN,
                                                      temperature_h[i * nv + lev],
                                                      pressure_h[i * nv + lev]);
                            Rd_h[i * nv + lev] = Rd_from_chi_H(chi_H);
                        }
                        else {
                            Rd_h[i * nv + lev] = sim.Rd;
                        }
                        it++;
                    }
                    if (ultrahot_thermo != NO_UH_THERMO) {
                        Cp_h[i * nv + lev] = Cp_from_chi_H(chi_H, temperature_h[i * nv + lev]);
                    }
                    else {
                        Cp_h[i * nv + lev] = sim.Cp;
                    }
                }
            }

            for (int lev = 0; lev < nv; lev++) {
                //              Density [kg/m3]
                Rho_h[i * nv + lev] =
                    pressure_h[i * nv + lev] / (temperature_h[i * nv + lev] * Rd_h[i * nv + lev]);

                //              Momentum [kg/m3 m/s]
                Mh_h[i * 3 * nv + 3 * lev + 0] = 0.0;
                Mh_h[i * 3 * nv + 3 * lev + 1] = 0.0;
                Mh_h[i * 3 * nv + 3 * lev + 2] = 0.0;

                //              Vertical momentum [kg/m3 m/s]
                W_h[i * nv + lev]        = 0.0; // Center of the layer.
                Wh_h[i * (nv + 1) + lev] = 0.0; // Layers interface.
            }
            Wh_h[i * (nv + 1) + nv] = 0.0;

            if (core_benchmark == ACOUSTIC_TEST) {
                // add density perturbation for acoustic wave test
                double dp, R, lambda0, phi0, vmode, r, g, f;
                double lat = lonlat_h[i * 2 + 1];
                double lon = lonlat_h[i * 2];
                vmode      = 1;         // vertical mode
                dp         = 100.0;     // pressure perturbation (Pa)
                R          = 1.0 / 3.0; // distance cutoff of perturbation
                lambda0    = 0;         //longitude of perturbation
                phi0       = 0;         //latitude of perturbation
                r          = acos(sin(phi0) * sin(lat) + cos(phi0) * cos(lat) * cos(lon - lambda0));
                if (r < R) {
                    f = 0.5 * (1 + cos(M_PI * r / R));
                }
                else {
                    f = 0.0;
                }
                for (int lev = 0; lev < nv; lev++) {
                    g = sin(vmode * M_PI * Altitude_h[lev] / sim.Top_altitude);

                    pressure_h[i * nv + lev] += dp * f * g;
                    Rho_h[i * nv + lev] =
                        pressure_h[i * nv + lev] / sim.Rd / temperature_h[i * nv + lev];
                }
            }
            else if (core_benchmark == GWAVE_TEST) {
                double dpt, R, lambda0, phi0, vmode, r, g, f;
                double lat   = lonlat_h[i * 2 + 1];
                double lon   = lonlat_h[i * 2];
                double kappa = sim.Rd / sim.Cp, pt;

                vmode   = 2;         // vertical mode
                dpt     = 10;        // potential temp perturbation (K)
                R       = 1.0 / 3.0; // distance cutoff of perturbation
                lambda0 = 0;         //longitude of perturbation
                phi0    = 0;         //latitude of perturbation
                r       = acos(sin(phi0) * sin(lat) + cos(phi0) * cos(lat) * cos(lon - lambda0));

                if (r < R) {
                    f = 0.5 * (1 + cos(M_PI * r / R));
                }
                else {
                    f = 0.0;
                }
                for (int lev = 0; lev < nv; lev++) {
                    g  = sin(vmode * M_PI * Altitude_h[lev] / sim.Top_altitude);
                    pt = temperature_h[i * nv + lev]
                         * pow(pressure_h[i * nv + lev] / sim.P_Ref, -kappa);

                    pt += dpt * f * g; // apply perturbation to potential temperature
                    temperature_h[i * nv + lev] =
                        pt * pow(pressure_h[i * nv + lev] / sim.P_Ref, kappa);
                    Rho_h[i * nv + lev] =
                        pressure_h[i * nv + lev] / (sim.Rd * temperature_h[i * nv + lev]);
                }
            }
        }
        if (core_benchmark == JET_STEADY) {
            //  Number of threads per block.
            const int NTH = 256;

            //  Specify the block sizes.
            dim3 NB((point_num / NTH) + 1, nv, 1);

            hipMemcpy(Altitude_d, Altitude_h, nv * sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(
                pressure_d, pressure_h, point_num * nv * sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(Mh_d, Mh_h, 3 * point_num * nv * sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(Rho_d, Rho_h, point_num * nv * sizeof(double), hipMemcpyHostToDevice);
            hipMemcpy(temperature_d,
                       temperature_h,
                       point_num * nv * sizeof(double),
                       hipMemcpyHostToDevice);
            hipMemcpy(lonlat_d, lonlat_h, 2 * point_num * sizeof(double), hipMemcpyHostToDevice);
            setup_jet<<<NB, NTH>>>(Mh_d,
                                   // setup_jet <<< 1, 1 >>>  (Mh_d,
                                   pressure_d,
                                   Rho_d,
                                   temperature_d,
                                   sim.Cp,
                                   sim.Rd,
                                   sim.Omega,
                                   sim.A,
                                   Altitude_d,
                                   lonlat_d,
                                   point_num);

            hipMemcpy(Mh_h, Mh_d, 3 * point_num * nv * sizeof(double), hipMemcpyDeviceToHost);
            hipMemcpy(temperature_h,
                       temperature_d,
                       point_num * nv * sizeof(double),
                       hipMemcpyDeviceToHost);
            hipMemcpy(
                pressure_h, pressure_d, point_num * nv * sizeof(double), hipMemcpyDeviceToHost);
            hipMemcpy(Rho_h, Rho_d, point_num * nv * sizeof(double), hipMemcpyDeviceToHost);
        }

        simulation_start_time = 0.0;
    } //end if rest
    else {
        bool load_OK = true;
        // build planet filename
        string planet_filename;

        path   p(initial_conditions_filename);
        int    file_number = 0;
        string basename    = "";

        string parent_path = p.parent();

        // Reload correct file if we are continuing from a specific file
        if (continue_sim) {
            if (!match_output_file_numbering_scheme(
                    initial_conditions_filename, basename, file_number)) {
                log::printf("Loading initial conditions: "
                            "Could not recognise file numbering scheme "
                            "for input %s: (found base: %s, num: %d) \n",
                            initial_conditions_filename.c_str(),
                            basename.c_str(),
                            file_number);
                return false;
            }

            output_file_idx = file_number;

            planet_filename = p.parent() + "/esp_output_planet_" + basename + ".h5";
        }
        else {
            planet_filename = p.parent() + "/" + p.stem() + "_planet.h5";
        }

        // check existence of files
        if (!path_exists(initial_conditions_filename)) {
            log::printf("initial condition file %s not found.\n",
                        initial_conditions_filename.c_str());
            return false;
        }

        if (!path_exists(planet_filename)) {
            log::printf("planet_file %s not found.\n", planet_filename.c_str());
            return false;
        }


        log::printf("Loading planet from: %s\n", planet_filename.c_str());
        log::printf("Loading initial conditions from: %s\n", initial_conditions_filename.c_str());

        // Check planet data
        {
            // values from initial conditions to check against variables from config
            map<string, double> mapValuesDouble;
            map<string, int>    mapValuesInt;

            mapValuesDouble["/A"]            = sim.A;
            mapValuesDouble["/Top_altitude"] = sim.Top_altitude;
            mapValuesInt["/glevel"]          = glevel;
            mapValuesInt["/vlevel"]          = nv;

            storage s(planet_filename, true);

            bool values_match = true;
            // double

            for (const std::pair<std::string, double> &element : mapValuesDouble) {
                double value = 0.0;
                load_OK      = s.read_value(element.first, value);

                if (!load_OK) {
                    printf("Error reading key %s from reload config.\n", element.first.c_str());
                    values_match = false;
                }


                if (value != element.second) {
                    log::printf("mismatch for %s value between config value: %f and initial "
                                "condition value %f.\n",
                                element.first.c_str(),
                                element.second,
                                value);
                    values_match = false;
                }
            }

            // int var
            for (const std::pair<std::string, int> &element : mapValuesInt) {
                int value = 0;
                load_OK   = s.read_value(element.first, value);

                if (!load_OK) {
                    printf("Error reading key %s from reload config.\n", element.first.c_str());
                    values_match = false;
                }

                if (value != element.second) {
                    log::printf("mismatch for %s value between config value: %d and initial "
                                "condition value %d.\n",
                                element.first.c_str(),
                                element.second,
                                value);
                    values_match = false;
                }
            }


            if (load_OK == false || values_match == false) {
                log::printf("Could not reload full configuration.\n");

                return false;
            }
        }


        //      Restart from an existing simulation.
        {
            // Load atmospheric data
            storage s(initial_conditions_filename, true);
            // Step number
            load_OK &= s.read_value("/nstep", nstep);

            log::printf("Reloaded %s: %d.\n", "/nstep", load_OK ? 1 : 0);

            //      Density
            load_OK &= s.read_table_to_ptr("/Rho", Rho_h, point_num * nv);
            log::printf("Reloaded %s: %d.\n", "/Rho", load_OK ? 1 : 0);
            //      Pressure
            load_OK &= s.read_table_to_ptr("/Pressure", pressure_h, point_num * nv);
            log::printf("Reloaded %s: %d.\n", "/Pressure", load_OK ? 1 : 0);
            //      Horizontal momentum
            load_OK &= s.read_table_to_ptr("/Mh", Mh_h, point_num * nv * 3);
            log::printf("Reloaded %s: %d.\n", "/Mh", load_OK ? 1 : 0);
            //      Vertical momentum
            load_OK &= s.read_table_to_ptr("/Wh", Wh_h, point_num * nvi);
            log::printf("Reloaded %s: %d.\n", "/Wh", load_OK ? 1 : 0);

            load_OK &= s.read_table_to_ptr("/Rd", Rd_h, point_num * nv);
            log::printf("Reloaded %s: %d.\n", "/Rd", load_OK ? 1 : 0);

            load_OK &= s.read_table_to_ptr("/Cp", Cp_h, point_num * nv);
            log::printf("Reloaded %s: %d.\n", "/Cp", load_OK ? 1 : 0);
            //      Simulation start time
            load_OK &= s.read_value("/simulation_time", simulation_start_time);
            log::printf("Reloaded %s: %d.\n", "/simulation_time", load_OK ? 1 : 0);
        }


        if (!load_OK) {
            log::printf("Error reloading simulation state\n");

            return false;
        }


        for (int i = 0; i < point_num; i++) {
            for (int lev = 0; lev < nv; lev++) {
                //hack
                // Rd_h[i * nv + lev] = sim.Rd;
                // Cp_h[i * nv + lev] = sim.Cp;
                ////
                temperature_h[i * nv + lev] =
                    pressure_h[i * nv + lev] / (Rd_h[i * nv + lev] * Rho_h[i * nv + lev]);
            }
        }


        for (int i = 0; i < point_num; i++) {
            for (int lev = 0; lev < nv; lev++) {
                double xi   = Altitude_h[lev];
                double xim1 = Altitudeh_h[lev];
                double xip1 = Altitudeh_h[lev + 1];

                double a = (xi - xip1) / (xim1 - xip1);
                double b = (xi - xim1) / (xip1 - xim1);

                W_h[i * nv + lev] = Wh_h[i * (nv + 1) + lev] * a + Wh_h[i * (nv + 1) + lev + 1] * b;
            }
        }
    } //end if rest == false
#ifdef BENCHMARKING
    // recompute temperature from pressure and density, to have correct rounding for binary comparison
    for (int i = 0; i < point_num; i++)
        for (int lev = 0; lev < nv; lev++)
            temperature_h[i * nv + lev] =
                pressure_h[i * nv + lev] / (Rd_h[i * nv + lev] * Rho_h[i * nv + lev]);
#endif // BENCHMARKING

    //  Diffusion
    //  Horizontal
    double *Kdhz_h, *Kdh4_h;
    Kdhz_h = new double[nv]; // horizontal divergence damping strength
    Kdh4_h = new double[nv]; // horizontal diffusion strength
                             // if (sim.DiffSponge) {
    double  n, ksponge;
    double *Kdh2_h;
    Kdh2_h = new double[nv];
    for (int lev = 0; lev < nv; lev++) {
        double dbar = sqrt(2 * M_PI / 5) * sim.A / (pow(2, glevel));
        Kdh4_h[lev] =
            (sim.Diffc) * pow(dbar, 4.) / timestep_dyn; // * Altitude_h[lev]/sim.Top_altitude;
        Kdhz_h[lev] =
            (sim.DivDampc) * pow(dbar, 4.) / timestep_dyn; // * Altitude_h[lev]/sim.Top_altitude;
        if (sim.DiffSponge) {
            n = Altitude_h[lev] / sim.Top_altitude;
            if (n > ns_diff_sponge) {
                ksponge = Dv_sponge
                          * pow(sin(0.5 * M_PI * (n - ns_diff_sponge) / (1.0 - ns_diff_sponge)), 2);
            }
            else {
                ksponge = 0;
            }
            if (order_diff_sponge == 2) {
                Kdh2_h[lev] = ksponge * pow(dbar, 2.) / timestep_dyn;
            }
            else if (order_diff_sponge == 4) {
                Kdh4_h[lev] += ksponge * pow(dbar, 4.) / timestep_dyn;
            }
        }
    }

    //  Diffusion
    //  Vertical
    double *Kdvz_h, *Kdv6_h;
    Kdvz_h = new double[nv]; // vertical divergence damping strength
    Kdv6_h = new double[nv]; // vertical diffusion strength
    for (int lev = 0; lev < nv; lev++) {
        //      Diffusion constant.
        // double dz   = sim.Top_altitude / nv;
        Kdv6_h[lev] = 0.0; //not used (yet? perhaps in future)
        Kdvz_h[lev] = 0.0; //not used (yet? perhaps in future)
    }


    //  Copy memory to the device
    hipMemcpy(point_local_d, point_local_h, 6 * point_num * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(maps_d,
               maps_h,
               (nl_region + 2) * (nl_region + 2) * nr * sizeof(int),
               hipMemcpyHostToDevice);
    hipMemcpy(Altitude_d, Altitude_h, nv * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Altitudeh_d, Altitudeh_h, nvi * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(nvecoa_d, nvecoa_h, 6 * 3 * point_num * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(nvecti_d, nvecti_h, 6 * 3 * point_num * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(nvecte_d, nvecte_h, 6 * 3 * point_num * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(areasTr_d, areasTr_h, 6 * point_num * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(areasT_d, areasT_h, point_num * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(areas_d, areas_h, 3 * 6 * point_num * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(lonlat_d, lonlat_h, 2 * point_num * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(func_r_d, func_r_h, 3 * point_num * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(
        temperature_d, temperature_h, point_num * nv * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Mh_d, Mh_h, point_num * nv * 3 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(W_d, W_h, point_num * nv * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Wh_d, Wh_h, point_num * nvi * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Rho_d, Rho_h, point_num * nv * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(pressure_d, pressure_h, point_num * nv * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(div_d, div_h, 7 * 3 * point_num * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(grad_d, grad_h, 7 * 3 * point_num * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Kdhz_d, Kdhz_h, nv * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Kdh4_d, Kdh4_h, nv * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Kdvz_d, Kdvz_h, nv * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Kdv6_d, Kdv6_h, nv * sizeof(double), hipMemcpyHostToDevice);

    hipMemcpy(Kdh2_d, Kdh2_h, nv * sizeof(double), hipMemcpyHostToDevice);


    if (sim.output_mean == true) {
        hipMemcpy(Mh_mean_d, Mh_h, point_num * nv * 3 * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(
            pressure_mean_d, pressure_h, point_num * nv * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(Wh_mean_d, Wh_h, point_num * nvi * sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(Rho_mean_d, Rho_h, point_num * nv * sizeof(double), hipMemcpyHostToDevice);
    }

    if (sim.RayleighSponge == true)
        hipMemcpy(zonal_mean_tab_d,
                   zonal_mean_tab_h,
                   3 * point_num * sizeof(int),
                   hipMemcpyHostToDevice);

    hipMemcpy(Rd_d, Rd_h, point_num * nv * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(Cp_d, Cp_h, point_num * nv * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(GibbsT_d, GibbsT, GibbsN * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(GibbsdG_d, GibbsdG, GibbsN * sizeof(double), hipMemcpyHostToDevice);

    //  Initialize arrays
    hipMemset(Adv_d, 0, sizeof(double) * 3 * point_num * nv);
    hipMemset(v_d, 0, sizeof(double) * nv * point_num * 3);
    hipMemset(pt_d, 0, sizeof(double) * nv * point_num);
    hipMemset(pth_d, 0, sizeof(double) * nvi * point_num);
    // hipMemset(pt_tau_d, 0, sizeof(double) * nv * point_num);
    hipMemset(epotential_d, 0, sizeof(double) * nv * point_num);
    hipMemset(epotentialh_d, 0, sizeof(double) * nvi * point_num);
    hipMemset(ekinetic_d, 0, sizeof(double) * nv * point_num);
    hipMemset(ekinetich_d, 0, sizeof(double) * nvi * point_num);
    hipMemset(Etotal_tau_d, 0, sizeof(double) * nv * point_num);

    hipMemset(SlowMh_d, 0, sizeof(double) * nv * point_num * 3);
    hipMemset(SlowWh_d, 0, sizeof(double) * nvi * point_num);
    hipMemset(SlowRho_d, 0, sizeof(double) * nv * point_num);
    hipMemset(Slowpressure_d, 0, sizeof(double) * nv * point_num);
    hipMemset(h_d, 0, sizeof(double) * nv * point_num);
    hipMemset(hh_d, 0, sizeof(double) * nvi * point_num);
    hipMemset(Rhos_d, 0, sizeof(double) * nv * point_num);
    hipMemset(pressures_d, 0, sizeof(double) * nv * point_num);
    hipMemset(Mhs_d, 0, sizeof(double) * nv * point_num * 3);
    hipMemset(Ws_d, 0, sizeof(double) * nv * point_num);
    hipMemset(Whs_d, 0, sizeof(double) * nvi * point_num);
    hipMemset(gtil_d, 0, sizeof(double) * nv * point_num);
    hipMemset(gtilh_d, 0, sizeof(double) * nvi * point_num);
    hipMemset(Rhok_d, 0, sizeof(double) * nv * point_num);
    hipMemset(pressurek_d, 0, sizeof(double) * nv * point_num);
    hipMemset(Mhk_d, 0, sizeof(double) * nv * point_num * 3);
    hipMemset(Wk_d, 0, sizeof(double) * nv * point_num);
    hipMemset(Whk_d, 0, sizeof(double) * nvi * point_num);
    hipMemset(Sp_d, 0, sizeof(double) * point_num * nv);
    hipMemset(Sd_d, 0, sizeof(double) * point_num * nv);
    hipMemset(DivM_d, 0, sizeof(double) * point_num * 3 * nv);
    hipMemset(diffpr_d, 0, sizeof(double) * nv * point_num);
    hipMemset(diffmh_d, 0, sizeof(double) * 3 * nv * point_num);
    hipMemset(diffw_d, 0, sizeof(double) * nv * point_num);
    hipMemset(diffrh_d, 0, sizeof(double) * nv * point_num);
    hipMemset(diff_d, 0, sizeof(double) * 6 * nv * point_num);
    hipMemset(divg_Mh_d, 0, sizeof(double) * 3 * nv * point_num);

    hipMemset(diffprv_d, 0, sizeof(double) * nv * point_num);
    hipMemset(diffmv_d, 0, sizeof(double) * 3 * nv * point_num);
    hipMemset(diffwv_d, 0, sizeof(double) * nv * point_num);
    hipMemset(diffrv_d, 0, sizeof(double) * nv * point_num);
    hipMemset(diffv_d1, 0, sizeof(double) * 6 * nv * point_num);
    hipMemset(diffv_d2, 0, sizeof(double) * 6 * nv * point_num);

    hipMemset(boundary_flux_d, 0, sizeof(double) * 6 * nv * point_num);

    delete[] Kdh4_h;
    delete[] Kdhz_h;
    delete[] Kdv6_h;
    delete[] Kdvz_h;
    delete[] Kdh2_h;

    // modules need to set their initial conditions
    if (phy_modules_execute) {
        if (sim.rest) // no initial condition file
            phy_modules_init_data(*this, sim, nullptr);
        else {
            // load initial condition file and pass it to modules
            storage s(initial_conditions_filename, true);

            phy_modules_init_data(*this, sim, &s);
        }
    }


    return true;
}

__host__ ESP::~ESP() {

    //
    //  Description: Frees the memory space.
    //
    //  Host
    // Simulation state data
    log::printf("Freeing ESP memory.\n");

    free(Rho_h);
    free(pressure_h);
    free(temperature_h);
    free(Mh_h);
    free(W_h);
    free(Wh_h);

    //  Device
    // Grid
    hipFree(point_local_d);
    hipFree(maps_d);

    //  Altitude (grid)
    hipFree(Altitude_d);
    hipFree(Altitudeh_d);

    //  Operators
    hipFree(nvecoa_d);
    hipFree(nvecti_d);
    hipFree(nvecte_d);
    hipFree(areasT_d);
    hipFree(areasTr_d);
    hipFree(areas_d);

    //  Longitude-latitude
    hipFree(lonlat_d);
    hipFree(div_d);
    hipFree(grad_d);
    hipFree(func_r_d);

    //  Temperature
    hipFree(temperature_d);
    //  Diagnostics
    hipFree(Mh_d);

    hipFree(W_d);
    hipFree(Wh_d);
    hipFree(Rho_d);
    hipFree(pressure_d);
    hipFree(pressureh_d);

    //  Entalphy
    hipFree(h_d);
    hipFree(hh_d);

    //  Advection
    hipFree(Adv_d);
    //  Effective gravity
    hipFree(gtil_d);
    hipFree(gtilh_d);
    //  3D vector
    hipFree(v_d);
    //  Potential temperature
    hipFree(pt_d);
    hipFree(pth_d);
    //  Slow modes
    hipFree(SlowMh_d);
    hipFree(SlowWh_d);
    hipFree(SlowRho_d);
    hipFree(Slowpressure_d);
    //  RK-Method
    hipFree(Rhok_d);
    hipFree(pressurek_d);
    hipFree(Mhk_d);
    hipFree(Whk_d);
    hipFree(Wk_d);
    //  Deviations
    hipFree(Rhos_d);
    hipFree(pressures_d);
    hipFree(Mhs_d);
    hipFree(Whs_d);
    hipFree(Ws_d);

    //  Vertical integration
    hipFree(Sd_d);
    hipFree(Sp_d);

    //  Diffusion
    hipFree(Kdhz_d);
    hipFree(Kdh4_d);
    hipFree(Kdvz_d);
    hipFree(Kdv6_d);
    hipFree(DivM_d);
    hipFree(diffpr_d);
    hipFree(diffmh_d);
    hipFree(diffw_d);
    hipFree(diffrh_d);
    hipFree(diff_d);
    hipFree(divg_Mh_d);

    hipFree(Kdh2_d);

    hipFree(diffprv_d);
    hipFree(diffmv_d);
    hipFree(diffwv_d);
    hipFree(diffrv_d);
    hipFree(diffv_d1);
    hipFree(diffv_d2);

    //  globdiag quantities
    hipFree(Etotal_d);
    hipFree(Entropy_d);
    hipFree(Mass_d);
    hipFree(AngMomx_d);
    hipFree(AngMomy_d);
    hipFree(AngMomz_d);
    hipFree(GlobalE_d);
    hipFree(GlobalEnt_d);
    hipFree(GlobalMass_d);
    hipFree(GlobalAMx_d);
    hipFree(GlobalAMy_d);
    hipFree(GlobalAMz_d);
    free(Etotal_h);
    free(Entropy_h);
    free(Mass_h);
    free(AngMomx_h);
    free(AngMomy_h);
    free(AngMomz_h);
    //  Extras-nan
    hipFree(check_d);

    // Sponge Layer
    hipFree(vbar_d);
    hipFree(zonal_mean_tab_d);
    hipFree(Tbar_d);

    free(vbar_h);
    free(utmp_h);
    free(vtmp_h);
    free(wtmp_h);

    free(Tbar_h);
    free(Ttmp_h);

    hipFree(utmp);
    hipFree(vtmp);
    hipFree(wtmp);
    hipFree(Ttmp);

    free(profx_Qheat_h);
    
    hipFree(profx_Qheat_d);
    hipFree(profx_dMh_d);
    hipFree(profx_dWh_d);
    hipFree(profx_dW_d);

    hipFree(epotential_d);
    hipFree(epotentialh_d);
    hipFree(ekinetic_d);
    hipFree(ekinetich_d);
    hipFree(Etotal_tau_d);

    // ultra hot
    free(Rd_h);
    free(Cp_h);
    free(GibbsT);
    free(GibbsdG);

    hipFree(Rd_d);
    hipFree(Cp_d);

    hipFree(GibbsT_d);
    hipFree(GibbsdG_d);

    hipFree(boundary_flux_d);
    free(boundary_flux_h);

    if (phy_modules_execute)
        phy_modules_free_mem();


    log::printf("\n\n Free memory!\n\n");
}
